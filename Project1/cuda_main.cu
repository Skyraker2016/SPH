#include "hip/hip_runtime.h"
// CUDA runtime �� + CUBLAS ��   
#include "hip/hip_runtime.h"  
#include "hipblas.h"  
#include<>
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <glut.h>
#include <vector>
#include <iostream>
#include <cmath>
using namespace std;

//����߽�
#define LEDGE -40
#define REDGE 40
#define LEVEL 50
#define LSIDE -40
#define RSIDE 40
#define FSIDE 40
#define BSIDE -40
#define USIDE 40
#define DSIDE -40
//������
#define xnum 16
#define ynum 16
#define znum 30


#define G -70.0
#define H 2.0
#define P0 1.0
#define K 50.0
#define PI 3.14159
#define M 1.0
#define U 200.0

#define BALL 3
#define PP 0.1

#define InitStep 1


#define t 0.1;


//typedef struct {
//	double x, y, z;
//	double vx, vy, vz;
//	double ax, ay, az;
//	double d;
//	double p;
//} Point;


void display();
void draw_point();

#define WIDTH 800
#define HEIGHT 800

float p_x[xnum*ynum*znum];
float p_y[xnum*ynum*znum];
float p_z[xnum*ynum*znum];
//float p_vx[xnum*ynum*znum];
//float p_vy[xnum*ynum*znum];
//float p_vz[xnum*ynum*znum];
//float p_ax[xnum*ynum*znum];
//float p_ay[xnum*ynum*znum];
//float p_az[xnum*ynum*znum];
float p_d[xnum*ynum*znum];
//float p_p[xnum*ynum*znum];

__global__ void kenelInitPoint(float *x, float *y, float *z, float *vx, float *vy, float *vz)
{

	int idx = threadIdx.x + blockIdx.x*ynum*znum;
	int i = idx / (ynum * znum);
	int j = idx / znum % ynum;
	int k = idx % znum;
	x[idx] = -InitStep*xnum / 2 + InitStep * i;
	y[idx] = -InitStep * xnum / 2 + InitStep * j;
	z[idx] = -InitStep * xnum / 2 + InitStep * k - 30;
	vx[idx] = 0;
	vy[idx] = 0;
	vz[idx] = 0;
	//printf(" xx %lf", z[idx]);
}


__global__ void kernelCountPoint_1(float *x, float *y, float *z, float *d, float *p) {
	int idx = threadIdx.x + blockIdx.x*ynum*znum;
	d[idx] = 0;
	for (int i = 0; i < xnum*ynum*znum; i++) {
		float dd = (x[idx] - x[i])*(x[idx] - x[i]) + (y[idx] - y[i])*(y[idx] - y[i]) + (z[idx] - z[i])*(z[idx] - z[i]);
		if (dd<H*H)
		{
			d[idx] += M * 315 * (H*H - dd)*(H*H - dd)*(H*H - dd) / (64 * PI*H*H*H*H*H*H*H*H*H);
		}
	}
	p[idx] = K * (d[idx] - P0);
}
__global__ void kernelCountPoint_2(float *x, float *y, float *z, float *vx, float *vy, float *vz, float *p, float *ax, float *ay, float *az) {
	int idx = threadIdx.x + blockIdx.x*ynum*znum;
	double yalix = 0, niandux = 0, yaliy = 0, nianduy = 0, yaliz = 0, nianduz = 0;
	for (int i = 0; i < xnum*ynum*znum; i++) {
		if (i != idx) {
			float d = (x[idx] - x[i])*(x[idx] - x[i]) + (y[idx] - y[i])*(y[idx] - y[i]) + (z[idx] - z[i])*(z[idx] - z[i]);
			if (d<H*H)
			{
				yaliz += (((p[idx] +p[i])* (z[idx] - z[i]) * ((H - sqrt(d))*(H - sqrt(d))) / (sqrt(d)  * (2 * p[idx]*p[i]))));
				nianduz += (vz[i] - vz[idx])  * (H - sqrt(d)) / (p[idx]*p[i]);

				yalix += (((p[idx] +p[i])* (x[idx] - x[i]) * ((H - sqrt(d))*(H - sqrt(d))) / (sqrt(d)  * (2 * p[idx]*p[i]))));
				niandux += (vx[i] - vx[idx])  * (H - sqrt(d)) / (p[idx]*p[i]);

				yaliy += (((p[idx] +p[i])* (y[idx] - y[i]) * ((H - sqrt(d))*(H - sqrt(d))) / (sqrt(d)  * (2 * p[idx]*p[i]))));
				nianduy += (vy[i] - vy[idx])  * (H - sqrt(d)) / (p[idx]*p[i]);

			}
		}
	}
	ax[idx] += 0.5 * M * 45 * yalix / (PI*H*H*H*H*H*H) + 0.2 * M * 45 * U*niandux / (PI*H*H*H*H*H*H);
	ay[idx] += 0.5 * M * 45 * yaliy / (PI*H*H*H*H*H*H) + 0.2 * M * 45 * U*nianduy / (PI*H*H*H*H*H*H);
	az[idx] += 0.5 * M * 45 * yaliz / (PI*H*H*H*H*H*H) + 0.2 * M * 45 * U*nianduz / (PI*H*H*H*H*H*H);
}
__global__ void kernelCountPoint_3(float *x, float *y, float *z, float *vx, float *vy, float *vz, float *p, float *ax, float *ay, float *az) {
	int idx = threadIdx.x + blockIdx.x*ynum*znum;
	double aax = ax[idx];
	double aay = ay[idx];
	double aaz = az[idx];
	aaz += G;
	vx[idx] = vx[idx] + aax * t;
	vy[idx] = vy[idx] + aay * t;
	vz[idx] = vz[idx] + aaz * t;


	x[idx] += vx[idx]*t;
	y[idx] += vy[idx]*t;
	z[idx] += vz[idx]*t;

	if (z[idx] < DSIDE ) {
		z[idx] = DSIDE;
		az[idx] = vz[idx] * (-2);
		vz[idx] = vz[idx] + aaz * t;
	}
	if (x[idx] < BSIDE || x[idx]>FSIDE) {
		x[idx] = x[idx] < BSIDE ? BSIDE : FSIDE;
		ax[idx] = vx[idx] * (-2);
		vx[idx] = vx[idx] + aax * t;
	}
	if (y[idx] < LSIDE || y[idx]>RSIDE) {
		y[idx] = y[idx] < LSIDE ? LSIDE : RSIDE;
		ay[idx] = vy[idx] * (-2);
		vy[idx] = vy[idx] + aay * t;
	}
}
float kd = 0;

int main()
{
	//int num = 0;
	//hipDeviceProp_t prop;
	//hipGetDeviceCount(&num);
	//for (int i = 0; i<num; i++)
	//{
	//	hipGetDeviceProperties(&prop, i);
	//}
	glfwInit();
	glfwWindowHint(GLFW_RESIZABLE, GL_FALSE);

	//��ʼ������
	GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "fluid", NULL, NULL);
	glfwMakeContextCurrent(window);


	//��ʼ��glew
	glewExperimental = GL_TRUE;
	glewInit();

	//������Ȳ���
	glEnable(GL_DEPTH_TEST);
	glDepthFunc(GL_LESS);


	float *dev_x = 0;
	float *dev_y = 0;
	float *dev_z = 0;
	float *dev_vx = 0;
	float *dev_vy = 0;
	float *dev_vz = 0;
	float *dev_ax = 0;
	float *dev_ay = 0;
	float *dev_az = 0;
	float *dev_d = 0;
	float *dev_p = 0;
	hipMalloc((void**)&dev_x, xnum*ynum*znum * sizeof(float));
	hipMalloc((void**)&dev_y, xnum*ynum*znum * sizeof(float));
	hipMalloc((void**)&dev_z, xnum*ynum*znum * sizeof(float));
	hipMalloc((void**)&dev_vx, xnum*ynum*znum * sizeof(float));
	hipMalloc((void**)&dev_vy, xnum*ynum*znum * sizeof(float));
	hipMalloc((void**)&dev_vz, xnum*ynum*znum * sizeof(float));
	hipMalloc((void**)&dev_ax, xnum*ynum*znum * sizeof(float));
	hipMalloc((void**)&dev_ay, xnum*ynum*znum * sizeof(float));
	hipMalloc((void**)&dev_az, xnum*ynum*znum * sizeof(float));
	hipMalloc((void**)&dev_d, xnum*ynum*znum * sizeof(float));
	hipMalloc((void**)&dev_p, xnum*ynum*znum * sizeof(float));
	kenelInitPoint << < xnum, ynum*znum >> > (dev_x, dev_y, dev_z, dev_vx, dev_vy, dev_vz);
	//����ɫ����
	glClearColor(1.0f, 1.0f, 1.0f, 1.0f);
	while (!glfwWindowShouldClose(window)) {
		//cout << "working..." << endl;
		kernelCountPoint_1 << < xnum, ynum*znum >> > (dev_x, dev_y, dev_z, dev_d, dev_p);
		kernelCountPoint_2 << < xnum, ynum*znum >> > (dev_x, dev_y, dev_z, dev_vx, dev_vy, dev_vz, dev_p, dev_ax, dev_ay, dev_az );
		kernelCountPoint_3 << < xnum, ynum*znum >> > (dev_x, dev_y, dev_z, dev_vx, dev_vy, dev_vz, dev_p, dev_ax, dev_ay, dev_az);
		hipMemcpy(p_x, dev_x, xnum*ynum*znum * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(p_y, dev_y, xnum*ynum*znum * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(p_z, dev_z, xnum*ynum*znum * sizeof(float), hipMemcpyDeviceToHost);
		display();
		glfwSwapBuffers(window);
		glfwPollEvents();
	}
	glfwTerminate();
	hipFree(dev_p);
	return 0;

}

//hipError_t initPoint(Point * dev_p) {
//	hipMemcpy(dev_p, point, xnum*ynum*znum * sizeof(Point), hipMemcpyHostToDevice);
//	kenelInitPoint <<<xnum, ynum*znum >>> (dev_p);
//	hipDeviceSynchronize();
//	hipMemcpy(point, dev_p, xnum*ynum*znum * sizeof(Point), hipMemcpyDeviceToHost);
//	return hipSuccess;
//}
//hipError_t countPoint(Point * dev_p) {
//
//	hipMemcpy(dev_p, point, xnum*ynum*znum * sizeof(Point), hipMemcpyHostToDevice);
//
//	kernelCountPoint_1 << <xnum, ynum*znum >> > (dev_p);
//	kernelCountPoint_2 << <xnum, ynum*znum >> > (dev_p);
//	kernelCountPoint_3 << <xnum, ynum*znum >> > (dev_p);
//
//	hipMemcpy(point, dev_p, xnum*ynum*znum * sizeof(Point), hipMemcpyDeviceToHost);
//	return hipSuccess;
//}

void display(){
    //�建��
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    //���ص�λ����
    glLoadIdentity();
    //͸�Ӿ���
    gluPerspective(60.0, 1, REDGE, LEDGE);
    //�������

     gluLookAt(150, 0, 0, 0, 0, 0, 0, 0, 1);

    //����ģʽ�����л���
    glColor3f(0.9, 0.6, 0.6);
    glBegin(GL_TRIANGLES);
    glVertex3f(LEDGE,LEDGE,LEDGE);
    glVertex3f(LEDGE,REDGE,LEDGE);
    glVertex3f(REDGE,REDGE,LEDGE);
        
    glVertex3f(REDGE,REDGE,LEDGE);
    glVertex3f(REDGE,LEDGE,LEDGE);
    glVertex3f(LEDGE,LEDGE,LEDGE);
    glEnd();

    glColor3f(0.0f, 0.0f, 0.0f);
    draw_point();
    //glColor3f(0.5f, 0.7f, 1.0f);
    //draw_face();
}
//
//void init(Point *dev_p){
//	initPoint(dev_p);
//	kd = (float)315 / (64 * PI*pow(H, 9));
//}

void draw_point(){
    for (int i=0; i<xnum; i++){
        for (int j=0; j<ynum; j++){
            for (int k=0; k<znum; k++){
				int idx = i * ynum*znum + j * znum + k;
				//cout << x[idx] << " " << y[idx] << " " << z[idx] << endl;
                glPointSize(2.0f);
                glBegin(GL_POINTS);
                glVertex3f(p_x[idx], p_y[idx], p_z[idx]);
                glEnd();
            }
        }
    }
}







